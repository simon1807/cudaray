
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

__global__ 
void cuda_run(uint32_t * img, int width) 
{
	int x = blockIdx.x;
	int y = blockIdx.y;	
	int r = (int)((255.0/200.0)*x);
	img[ y * width + x ] = 0xff000000 | (r << 16);
}
 
void cuda_main( int width, int height, uint32_t * img )
{
    uint32_t * cuda_img;
    const int size = width * height * sizeof( uint32_t );
    
    hipMalloc( (void **)&cuda_img, size );
    hipMemset( cuda_img, 0, size );
    
    dim3 dimBlock( 1, 1 );
	dim3 dimGrid( width, height );
	
	cuda_run<<<dimGrid, dimBlock>>>( cuda_img, width );
	
	hipMemcpy( img, cuda_img, size, hipMemcpyDeviceToHost ); 
	hipFree( cuda_img );
}
