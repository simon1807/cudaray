#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#include "cudaray.h"
#include "mathlib.h"

#ifndef __HIPCC__ /* Ghetto CUDA. */
    #define __device__
    #define __global__

    #include <stdlib.h>
    #include <string.h>

    static void hipMalloc( void * p, size_t size )
    {
        void ** out = (void **)p;
        *out = malloc( size );
    }

    static void hipMemset( void * ptr, int x, size_t size )
    {
        memset( ptr, x, size );
    }

    static void hipMemcpy( void * out, const void * in, size_t size, int direction )
    {
        memcpy( out, in, size );
    }

    static void hipFree( void * ptr )
    {
        free( ptr );
    }

    #define hipMemcpyHostToDevice 0
    #define hipMemcpyDeviceToHost 0

    static struct
    {
        int x;
        int y;
    } blockIdx;
#endif

struct t_ray
{
    t_vec3 start;
    t_vec3 direction;
};

__device__
static inline float pow2( float x )
{
    return x * x;
}

__device__
static int sphere_intersect( const t_ray ray, const t_sphere * sphere, t_vec3 out, float * out_distance )
{
    float r = sphere->radius;

    t_vec3 d;
    vec3_dup( d, ray.start );
    vec3_sub( d, sphere->position );

    float a = vec3_dot( ray.direction, ray.direction );
    float b = 2 * vec3_dot( d, ray.direction );
    float c = vec3_dot( d, d ) - pow2(r);
    float delta = pow2(b) - 4 * a * c;

    if( delta < 0.0f )
        return 0;

    float t;

    if( b > 0 )
        t = (-b + sqrtf( delta )) / (2 * a);
    else
        t = (-b - sqrtf( delta )) / (2 * a);

    vec3_dup( out, ray.direction );
    vec3_scale( out, t );
    vec3_add( out, ray.start );
    *out_distance = vec3_dist( ray.start, out );

    return 1;
}

__global__ 
void cuda_run( uint32_t * img, int width, t_sphere * sphere_array, int sphere_count, t_light * light_array, int light_count )
{
	int x = blockIdx.x;
	int y = blockIdx.y;

    t_ray ray;
    vec3_set( ray.start, x, y, 1000.0f );
    vec3_set( ray.direction, 0.0f, 0.0f, 1.0f );

    t_sphere * best_sphere = NULL;
    float best_distance = 0.0f;
    t_vec3 best_intersect_point;

    for( int i = 0; i < sphere_count; ++i )
    {
        t_sphere * sphere = &sphere_array[i];

        t_vec3 intersect_point;
        float distance;
        if( !sphere_intersect( ray, sphere, intersect_point, &distance ) )
            continue;

        if( best_sphere == NULL || best_distance > distance )
        {
            best_sphere = sphere;
            best_distance = distance;
            vec3_dup( best_intersect_point, intersect_point );
        }
    }

    if( best_sphere == NULL )
        return;

    t_vec3 fragment_color;
    vec3_zero( fragment_color );

    for( int n_light = 0; n_light < light_count; ++n_light )
    {
        t_light * light = &light_array[n_light];

        t_vec3 light_vector;
        vec3_direction( light_vector, best_intersect_point, light->position );

        t_vec3 normal;
        vec3_direction( normal, best_sphere->position, best_intersect_point );

        float intensity = vec3_dot( normal, light_vector );
        if( intensity < 0 )
            continue;

        intensity *= light->intensity;

        vec3_dup( ray.start, best_intersect_point );
        vec3_direction( ray.direction, best_intersect_point, light->position );

        bool unobstructed = true;
        for( int j = 0; j < sphere_count; ++j )
        {
            t_sphere * sphere = &sphere_array[j];
            if( sphere == best_sphere )
                continue;

            float distance;
            t_vec3 point;
            if( sphere_intersect( ray, sphere, point, &distance ) )
            {
                unobstructed = false;
                break;
            }
        }

        if( !unobstructed )
            continue;

        t_vec3 light_color;
        vec3_dup( light_color, light->color );
        vec3_scale( light_color, intensity );

        t_vec3 color;
        vec3_dup( color, best_sphere->color );
        vec3_scalar_mul( color, light_color );
        vec3_clamp( color, 0.0f, 1.0f );

        vec3_add( fragment_color, color );
    }

    vec3_clamp( fragment_color, 0.0f, 1.0f );
    int r = 255 * fragment_color[0];
    int g = 255 * fragment_color[1];
    int b = 255 * fragment_color[2];
    img[ y * width + x ] += 0xff000000 | (r << 16) | (g << 8) | (b);
}
 
void cuda_main( int width, int height, uint32_t * img, t_sphere * sphere_array, int sphere_count, t_light * light_array, int light_count )
{
    uint32_t * cuda_img;
    t_sphere * cuda_sphere_array;
    t_light * cuda_light_array;
    const int size = width * height * sizeof( uint32_t );
    
    hipMalloc( &cuda_img, size );
    hipMalloc( &cuda_sphere_array, sphere_count * sizeof( t_sphere ) );
    hipMalloc( &cuda_light_array, light_count * sizeof( t_light ) );
    hipMemset( cuda_img, 0, size );
    hipMemcpy( cuda_sphere_array, sphere_array, sphere_count * sizeof( t_sphere ), hipMemcpyHostToDevice );
    hipMemcpy( cuda_light_array, light_array, light_count * sizeof( t_light ), hipMemcpyHostToDevice );

    #ifdef __HIPCC__
        dim3 dimBlock( 1, 1 );
        dim3 dimGrid( width, height );
	
        cuda_run<<<dimGrid, dimBlock>>>( cuda_img, width, cuda_sphere_array, sphere_count, cuda_light_array, light_count );
    #else
        for( int y = 0; y < height; ++y )
        {
            for( int x = 0; x < width; ++x )
            {
                blockIdx.x = x;
                blockIdx.y = y;
                cuda_run( cuda_img, width, cuda_sphere_array, sphere_count, cuda_light_array, light_count );
            }
        }
    #endif
	
	hipMemcpy( img, cuda_img, size, hipMemcpyDeviceToHost ); 
	hipFree( cuda_img );
    hipFree( cuda_sphere_array );
    hipFree( cuda_light_array );
}
